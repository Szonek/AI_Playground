
#include <hip/hip_runtime.h>
__global__ void vec_add(float *A, float *B, float *C)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
	for(int i = 0; i < 1024; i++)
	{
	    C[idx] += (A[idx] + B[idx]);
	}
}